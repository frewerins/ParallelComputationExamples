
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cassert>


#define BLOCKSIZE 256

__global__ void Difference(int n, int* input, int* result) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int x_i = input[tid];
    if (tid > 0) {
        int x_i_minus = input[tid - 1];

        result[tid] = x_i - x_i_minus;
    } else {
        result[tid] = x_i;
    }
}


int main() {
    int N = 1 << 28;

    int* h_array = new int[N];
    int* h_diff = new int[N];
    for (int i = 0; i < N; ++i) {
        h_array[i] = i + 1;
    }
    
    int* d_array;
    int* d_diff;
    unsigned int size = N * sizeof(int);
    hipMalloc(&d_array, size);
    hipMalloc(&d_diff, size);

    hipMemcpy(d_array, h_array, size, hipMemcpyHostToDevice);
    
    int num_blocks = (N + BLOCKSIZE - 1) / BLOCKSIZE;

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    Difference<<<num_blocks, BLOCKSIZE>>>(N, d_array, d_diff);
    hipEventRecord(stop);


    hipMemcpy(h_diff, d_diff, size, hipMemcpyDeviceToHost);

    float milliseconds;
    hipEventSynchronize(stop);


    hipEventElapsedTime(&milliseconds, start, stop);

    for (int i = 0; i < N; ++i) {
        assert(h_diff[i] == 1);
    }

    std::cout << milliseconds << " elapsed" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_array);
    hipFree(d_diff);
    delete[] h_array;
    delete[] h_diff;

}
